#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2016, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include "../Helpers.h"
#include "../Random.h"

using namespace optix;

struct PerRayData_radiance
{
    float3 result;
    float  importance;
    int    depth;
};

rtDeclareVariable(float3,        eye, , );
rtDeclareVariable(float3,        U, , );
rtDeclareVariable(float3,        V, , );
rtDeclareVariable(float3,        W, , );
rtDeclareVariable(float3,        bad_color, , );
rtDeclareVariable(float,         scene_epsilon, , );
rtBuffer<uchar4, 2>              output_buffer;
rtBuffer<float4, 2>              accum_buffer;
rtDeclareVariable(rtObject,      top_object, , );
rtDeclareVariable(unsigned int,  radiance_ray_type, , );
rtDeclareVariable(unsigned int,  frame, , );
rtDeclareVariable(uint2,         launch_index, rtLaunchIndex, );

rtDeclareVariable(float, aperture_radius, , );
rtDeclareVariable(float, focal_scale, , );
rtDeclareVariable(float4, jitter4, , );

#define NB_CLIP_PLANES 6
rtDeclareVariable(float4, clip_plane1, , );
rtDeclareVariable(float4, clip_plane2, , );
rtDeclareVariable(float4, clip_plane3, , );
rtDeclareVariable(float4, clip_plane4, , );
rtDeclareVariable(float4, clip_plane5, , );
rtDeclareVariable(float4, clip_plane6, , );

__device__ void getClippingValues(
        const float3& ray_origin, const float3& ray_direction,
        float& near, float& far )
{
    float4 clip_planes[ NB_CLIP_PLANES ] = {
        clip_plane1, clip_plane2, clip_plane3, clip_plane4, clip_plane5, clip_plane6
    };
    for( int i = 0; i < NB_CLIP_PLANES; ++i )
    {
        const float3 planeNormal = { clip_planes[i].x, clip_planes[i].y, clip_planes[i].z };
        float rn = dot( ray_direction, planeNormal );
        if( rn == 0.f )
            rn = scene_epsilon;
        float d = clip_planes[i].w;
        float t = -( dot( planeNormal, ray_origin ) + d ) / rn;
        if( rn > 0.f ) // opposite direction plane
            near = max( near, t );
        else
            far = min( far, t );
    }
}

RT_PROGRAM void camera()
{
    size_t2 screen = output_buffer.size();
    unsigned int seed =
        tea< 16 >( screen.x * launch_index.y + launch_index.x, frame );

    // Subpixel jitter: send the ray through a different position inside the pixel each time,
    // to provide antialiasing.
    float2 subpixel_jitter =
        frame == 0 ?
        make_float2(0.0f, 0.0f) :
        make_float2( rnd( seed ) - 0.5f, rnd( seed ) - 0.5f );

    float2 d = (make_float2(launch_index) + subpixel_jitter) / make_float2(screen) * 2.f - 1.f;

    float3 ray_origin = eye;
    float3 ray_direction = d.x * U + d.y * V + W;

    float fs = focal_scale == 0.f ? 1.f : focal_scale;
    float3 ray_target = ray_origin + fs * ray_direction;

    // lens sampling
    float2 sample =
        optix::square_to_disk( make_float2( jitter4.z, jitter4.w ));

    ray_origin =
        ray_origin + aperture_radius *
        ( sample.x * normalize( U ) +  sample.y * normalize( V ) );

    ray_direction = normalize( ray_target - ray_origin );

    float near = scene_epsilon;
    float far = INFINITY;
    getClippingValues( ray_origin, ray_direction, near, far );
    optix::Ray ray( ray_origin, ray_direction, radiance_ray_type, near, far );

    PerRayData_radiance prd;
    prd.importance = 1.f;
    prd.depth = 0;

    rtTrace( top_object, ray, prd );

    float4 acc_val = accum_buffer[ launch_index ];

    if( frame > 0 )
        acc_val = lerp(
            acc_val,
            make_float4( prd.result, 0.f), 1.0f / static_cast<float>( frame+1 ));
    else
        acc_val = make_float4( prd.result, 0.f );

    output_buffer[ launch_index ] = make_color( make_float3( acc_val ) );
    accum_buffer[ launch_index ] = acc_val;
}

RT_PROGRAM void exception()
{
    output_buffer[ launch_index ] = make_color( bad_color );
}

