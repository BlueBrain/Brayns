#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2016, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <optix_world.h>

struct PerRayData_radiance
{
    float3 result;
    float importance;
    int depth;
};

rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(float3, bg_color, , );
rtTextureSampler<float4, 2> envmap;
rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );

//
// Constant background
//
RT_PROGRAM void miss()
{
    prd_radiance.result = bg_color;
}

//
// Environment map background
//
RT_PROGRAM void envmap_miss()
{
    float theta = atan2f( ray.direction.x, ray.direction.z );
    float phi   = M_PIf * 0.5f -  acosf( ray.direction.y );
    float u     = (theta + M_PIf) * (0.5f * M_1_PIf);
    float v     = -0.5f * ( 1.0f + sin(phi) );
    prd_radiance.result = make_float3( tex2D(envmap, u, v) );
}
