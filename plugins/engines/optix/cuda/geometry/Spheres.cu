#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2016, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/*
 * Cyrille Favreau <cyrille.favreau@epfl.ch>
 * Added support for multiple spheres
 */

#include <optix_world.h>

using namespace optix;

rtBuffer<float> spheres;

rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, ); 
rtDeclareVariable(float3, shading_normal, attribute shading_normal, ); 
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(unsigned int, sphere_size, ,);

// Global variables
rtDeclareVariable(float, timestamp, , );

template<bool use_robust_method>
static __device__
void intersect_sphere( int primIdx )
{
    const int idx = primIdx * sphere_size;
    const float ts = spheres[ idx + 4 ];
    if( ts >= timestamp )
        return;

    const float3 center = { spheres[ idx ], spheres[ idx + 1 ], spheres[ idx + 2 ] };
    const float3 O = ray.origin - center;
    const float3 D = ray.direction;
    const float radius = spheres[ idx + 3 ];

    float b = dot( O, D );
    float c = dot( O, O ) - radius * radius;
    float disc = b * b - c;
    if( disc > 0.0f )
    {
        float sdisc = sqrtf( disc );
        float root1 = ( -b - sdisc );

        bool do_refine = false;

        float root11 = 0.0f;

        if( use_robust_method && fabsf(root1) > 10.f * radius )
            do_refine = true;

        if( do_refine )
        {
            // refine root1
            float3 O1 = O + root1 * ray.direction;
            b = dot(O1, D);
            c = dot(O1, O1) - radius*radius;
            disc = b*b - c;

            if( disc > 0.0f )
            {
                sdisc = sqrtf( disc );
                root11 = ( -b - sdisc );
            }
        }

        bool check_second = true;
        if( rtPotentialIntersection( root1 + root11 ) )
        {
            shading_normal = geometric_normal = (O + (root1 + root11)*D)/radius;
            if( rtReportIntersection( 0 ))
                check_second = false;
        }
        if(check_second)
        {
            float root2 = (-b + sdisc) + (do_refine ? root1 : 0);
            if( rtPotentialIntersection( root2 ) )
            {
                shading_normal = geometric_normal = (O + root2*D)/radius;
                rtReportIntersection( 0 );
            }
        }
    }
}


RT_PROGRAM void intersect( int primIdx )
{
    intersect_sphere<false>( primIdx );
}


RT_PROGRAM void robust_intersect( int primIdx )
{
    intersect_sphere<true>( primIdx );
}


RT_PROGRAM void bounds( int primIdx, float result[6] )
{
    const int idx = primIdx * sphere_size;
    const float3 cen = { spheres[ idx ], spheres[ idx + 1 ], spheres[ idx + 2 ] };
    const float3 rad = make_float3( spheres[ idx + 3 ] );

    optix::Aabb* aabb = (optix::Aabb*)result;

    if( rad.x > 0.0f  && !isinf(rad.x) )
    {
        aabb->m_min = cen - rad;
        aabb->m_max = cen + rad;
    }
    else
    {
        aabb->invalidate();
    }
}

