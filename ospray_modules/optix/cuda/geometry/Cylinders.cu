#include "hip/hip_runtime.h"
/* Copyright (c) 2015-2016, EPFL/Blue Brain Project
 * All rights reserved. Do not distribute without permission.
 * Responsible Author: Cyrille Favreau <cyrille.favreau@epfl.ch>
 *
 * This file is part of Brayns <https://github.com/BlueBrain/Brayns>
 *
 * This library is free software; you can redistribute it and/or modify it under
 * the terms of the GNU Lesser General Public License version 3.0 as published
 * by the Free Software Foundation.
 *
 * This library is distributed in the hope that it will be useful, but WITHOUT
 * ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS
 * FOR A PARTICULAR PURPOSE.  See the GNU Lesser General Public License for more
 * details.
 *
 * You should have received a copy of the GNU Lesser General Public License
 * along with this library; if not, write to the Free Software Foundation, Inc.,
 * 51 Franklin Street, Fifth Floor, Boston, MA 02110-1301 USA.
 */

#include <optix_world.h>

using namespace optix;

// Global variables
rtDeclareVariable(float, timestamp, , );

rtBuffer<float> cylinders;

// Geometry specific variables
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(unsigned int, cylinder_size, ,);

template<bool use_robust_method>
static __device__
void intersect_cylinder( int primIdx )
{
    const int idx = primIdx * cylinder_size;
    const float ts = cylinders[ idx + 7 ];
    if( ts >= timestamp )
        return;

    const float3 v0 = { cylinders[ idx ], cylinders[ idx + 1 ], cylinders[ idx + 2 ] };
    const float3 v1 = { cylinders[ idx + 3 ], cylinders[ idx + 4 ], cylinders[ idx + 5 ] };
    const float radius = cylinders[ idx + 6 ];

    const float3 A = v0 - ray.origin;
    const float3 B = v1 - ray.origin;

    const float3 O = make_float3( 0.f );
    const float3 V = ray.direction;

    const float3 AB = B - A;
    const float3 AO = O - A;

    const float3 AOxAB = cross( AO, AB );
    const float3 VxAB = cross( V, AB );
    const float ab2 = dot( AB, AB );
    const float a = dot( VxAB, VxAB );
    const float b = 2.f * dot( VxAB, AOxAB );
    const float c = dot( AOxAB, AOxAB ) - ( radius * radius * ab2 );

    const float radical = b * b - 4.f * a * c;
    if( radical >= 0.f )
    {
        // clip to near and far cap of cylinder
        const float tA = dot( AB, A ) / dot( V, AB );
        const float tB = dot( AB, B ) / dot( V, AB );
        //const float tAB0 = max( 0.f, min( tA, tB ));
        //const float tAB1 = min( RT_DEFAULT_MAX, max( tA, tB ));
        const float tAB0 = min( tA, tB );
        const float tAB1 = max( tA, tB );

        const float srad = sqrt( radical );

        const float t_in = ( -b - srad ) / ( 2.f * a );

        bool check_second = true;
        if( t_in >= tAB0 && t_in <= tAB1 )
        {
            if( rtPotentialIntersection( t_in ))
            {
                const float3 P = ray.origin + t_in * ray.direction - v0;
                const float3 V = cross( P, AB );
                geometric_normal = shading_normal = cross( AB, V );
                if( rtReportIntersection( 0 ))
                    check_second = false;
            }
        }

        if( check_second )
        {
            const float t_out= ( -b + srad ) / ( 2.f * a );
            if( t_out >= tAB0 && t_out <= tAB1 )
            {
                if( rtPotentialIntersection( t_out ))
                {
                    const float3 P = t_out * ray.direction - A;
                    const float3 V = cross( P, AB);
                    geometric_normal = shading_normal = cross( AB, V );
                    rtReportIntersection( 0 );
                }
            }
        }
    }
}

RT_PROGRAM void intersect( int primIdx )
{
    intersect_cylinder<false>( primIdx );
}

RT_PROGRAM void robust_intersect( int primIdx )
{
    intersect_cylinder<true>( primIdx );
}

RT_PROGRAM void bounds( int primIdx, float result[6] )
{
    const int idx = primIdx * cylinder_size;
    const float3 v0 = { cylinders[ idx ], cylinders[ idx + 1 ], cylinders[ idx + 2 ] };
    const float3 v1 = { cylinders[ idx + 3 ], cylinders[ idx + 4 ], cylinders[ idx + 5 ] };
    const float radius = cylinders[ idx + 6 ];

    optix::Aabb* aabb = (optix::Aabb*)result;

    if( radius > 0.f  && !isinf( radius ))
    {
        aabb->m_min = fminf(v0,v1) - radius;
        aabb->m_max = fmaxf(v0,v1) + radius;
    }
    else
        aabb->invalidate();
}
