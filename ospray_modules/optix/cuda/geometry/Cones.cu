#include "hip/hip_runtime.h"
/* Copyright (c) 2015-2017, EPFL/Blue Brain Project
 * All rights reserved. Do not distribute without permission.
 * Responsible Author: Cyrille Favreau <cyrille.favreau@epfl.ch>
 * Author: Jafet Villafranca Diaz <jafet.villafrancadiaz@epfl.ch>
 *
 * Ray-cone intersection:
 * based on Ching-Kuang Shene (Graphics Gems 5, p. 227-230)
 *
 * This file is part of Brayns <https://github.com/BlueBrain/Brayns>
 *
 * This library is free software; you can redistribute it and/or modify it under
 * the terms of the GNU Lesser General Public License version 3.0 as published
 * by the Free Software Foundation.
 *
 * This library is distributed in the hope that it will be useful, but WITHOUT
 * ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS
 * FOR A PARTICULAR PURPOSE.  See the GNU Lesser General Public License for more
 * details.
 *
 * You should have received a copy of the GNU Lesser General Public License
 * along with this library; if not, write to the Free Software Foundation, Inc.,
 * 51 Franklin Street, Fifth Floor, Boston, MA 02110-1301 USA.
 */

#include <optix_world.h>

using namespace optix;

// Global variables
rtDeclareVariable(float, timestamp, , );
rtDeclareVariable(unsigned int, cone_size, , );

rtBuffer<float> cones;

// Geometry specific variables
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );

template <bool use_robust_method>
static __device__ void intersect_cone(int primIdx)
{
    const int idx = primIdx * cone_size;
    const float ts = cones[idx + 8];
    if (ts > 0 && timestamp > ts)
        return;

    float3 v0 = {cones[idx], cones[idx + 1], cones[idx + 2]};
    float3 v1 = {cones[idx + 3], cones[idx + 4], cones[idx + 5]};
    float radius0 = cones[idx + 6];
    float radius1 = cones[idx + 7];

    if (radius0 < radius1)
    {
        // swap radii and positions, so radius0 and v0 are always at the bottom
        float tmpRadius = radius1;
        radius1 = radius0;
        radius0 = tmpRadius;

        float3 tmpPos = v1;
        v1 = v0;
        v0 = tmpPos;
    }

    const float3 upVector = v1 - v0;
    const float upLength = length(upVector);

    // Compute the height of the full cone, in order to obtain its vertex
    const float deltaRadius = radius0 - radius1;
    const float tanA = deltaRadius / upLength;
    const float coneHeight = radius0 / tanA;
    const float squareTanA = tanA * tanA;
    const float div = sqrtf(1.f + squareTanA);
    if (div == 0.f)
        return;

    const float cosA = 1.f / div;

    const float3 V = v0 + normalize(upVector) * coneHeight;
    const float3 v = normalize(v0 - V);

    // Normal of the plane P determined by V and ray
    float3 n = normalize(cross(ray.direction, V - ray.origin));
    const float dotNV = dot(n, v);
    if (dotNV > 0.f)
        n = n * -1.f;

    const float squareCosTheta = 1.f - dotNV * dotNV;
    const float cosTheta = sqrtf(squareCosTheta);
    if (cosTheta < cosA)
        return; // no intersection

    if (squareCosTheta == 0.f)
        return;

    const float squareTanTheta = (1.f - squareCosTheta) / squareCosTheta;
    const float tanTheta = sqrtf(squareTanTheta);

    // Compute u-v-w coordinate system
    const float3 u = normalize(cross(v, n));
    const float3 w = normalize(cross(u, v));

    // Circle intersection of cone with plane P
    const float3 uComponent = sqrtf(squareTanA - squareTanTheta) * u;
    const float3 vwComponent = v + tanTheta * w;
    const float3 delta1 = vwComponent + uComponent;
    const float3 delta2 = vwComponent - uComponent;
    const float3 rayApex = V - ray.origin;

    const float3 normal1 = cross(ray.direction, delta1);
    const float length1 = length(normal1);

    if (length1 == 0.f)
        return;

    const float r1 = dot(cross(rayApex, delta1), normal1) / (length1 * length1);

    const float3 normal2 = cross(ray.direction, delta2);
    const float length2 = length(normal2);

    if (length2 == 0.f)
        return;

    const float r2 = dot(cross(rayApex, delta2), normal2) / (length2 * length2);

    float t_in = r1;
    float t_out = r2;
    if (r2 > 0.f)
    {
        if (r1 > 0.f)
        {
            if (r1 > r2)
            {
                t_in = r2;
                t_out = r1;
            }
        }
        else
            t_in = r2;
    }

    bool check_second = true;
    if (t_in > 0.f)
    {
        const float3 p1 = ray.origin + t_in * ray.direction;
        // consider only the parts within the extents of the truncated cone
        if (dot(p1 - v1, v) > 0.f && dot(p1 - v0, v) < 0.f)
        {
            if (rtPotentialIntersection(t_in))
            {
                const float3 surfaceVec = normalize(p1 - V);
                geometric_normal = shading_normal =
                    cross(cross(v, surfaceVec), surfaceVec);
                if (rtReportIntersection(0))
                    check_second = false;
            }
        }
    }

    if (check_second)
    {
        if (t_out > 0.f)
        {
            const float3 p2 = ray.origin + t_out * ray.direction;
            // consider only the parts within the extents of the truncated cone
            if (dot(p2 - v1, v) > 0.f && dot(p2 - v0, v) < 0.f)
            {
                if (rtPotentialIntersection(t_out))
                {
                    const float3 surfaceVec = normalize(p2 - V);
                    geometric_normal = shading_normal =
                        cross(cross(v, surfaceVec), surfaceVec);
                    rtReportIntersection(0);
                }
            }
        }
    }
}

RT_PROGRAM void intersect(int primIdx)
{
    intersect_cone<false>(primIdx);
}

RT_PROGRAM void robust_intersect(int primIdx)
{
    intersect_cone<true>(primIdx);
}

RT_PROGRAM void bounds(int primIdx, float result[6])
{
    const int idx = primIdx * cone_size;
    const float3 v0 = {cones[idx], cones[idx + 1], cones[idx + 2]};
    const float3 v1 = {cones[idx + 3], cones[idx + 4], cones[idx + 5]};
    const float radius = max(cones[idx + 6], cones[idx + 7]);

    const float3 V0 = {min(v0.x, v1.x), min(v0.y, v1.y), min(v0.z, v1.z)};
    const float3 V1 = {max(v0.x, v1.x), max(v0.y, v1.y), max(v0.z, v1.z)};

    optix::Aabb* aabb = (optix::Aabb*)result;

    if (radius > 0.0f && !isinf(radius))
    {
        aabb->m_min = V0 - radius;
        aabb->m_max = V1 + radius;
    }
    else
        aabb->invalidate();
}
