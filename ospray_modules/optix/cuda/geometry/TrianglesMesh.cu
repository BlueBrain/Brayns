#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2016, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/*
 * Cyrille Favreau <cyrille.favreau@epfl.ch>
 * Adaptations to Brayns
 */

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_matrix_namespace.h>
#include <optixu/optixu_aabb_namespace.h>
#include "IntersectionRefinement.h"

using namespace optix;

// This is to be plugged into an RTgeometry object to represent
// a triangle mesh with a vertex buffer of triangle soup (triangle list)
// with an interleaved position, normal, texturecoordinate layout.

rtBuffer<float3> vertices_buffer;
rtBuffer<float3> normal_buffer;
rtBuffer<float2> texcoord_buffer;
rtBuffer<int3>   indices_buffer;

rtDeclareVariable(float3, texcoord,         attribute texcoord, );
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3, shading_normal,   attribute shading_normal, );

rtDeclareVariable(float3, back_hit_point,   attribute back_hit_point, );
rtDeclareVariable(float3, front_hit_point,  attribute front_hit_point, );

rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );


template<bool DO_REFINE>
static __device__
void meshIntersect( int primIdx )
{
  const int3 v_idx = indices_buffer[primIdx];

  const float3 p0 = vertices_buffer[ v_idx.x ];
  const float3 p1 = vertices_buffer[ v_idx.y ];
  const float3 p2 = vertices_buffer[ v_idx.z ];

  // Intersect ray with triangle
  float3 n;
  float  t, beta, gamma;
  if( intersect_triangle( ray, p0, p1, p2, n, t, beta, gamma ) )
  {
    if(  rtPotentialIntersection( t ) )
    {
      geometric_normal = normalize( n );
      if( normal_buffer.size() == 0 )
        shading_normal = geometric_normal;
      else
      {
        float3 n0 = normal_buffer[ v_idx.x ];
        float3 n1 = normal_buffer[ v_idx.y ];
        float3 n2 = normal_buffer[ v_idx.z ];
        shading_normal = normalize( n1 * beta + n2 * gamma + n0 * ( 1.f - beta-gamma ));
      }

      if( texcoord_buffer.size() == 0 )
        texcoord = make_float3( 0.f, 0.f, 0.f );
      else {
        float2 t0 = texcoord_buffer[ v_idx.x ];
        float2 t1 = texcoord_buffer[ v_idx.y ];
        float2 t2 = texcoord_buffer[ v_idx.z ];
        texcoord = make_float3( t1 * beta + t2 * gamma + t0 * ( 1.f - beta-gamma ));
      }

      if( DO_REFINE )
          refine_and_offset_hitpoint(
                  ray.origin + t*ray.direction,
                  ray.direction,
                  geometric_normal,
                  p0,
                  back_hit_point,
                  front_hit_point );

      rtReportIntersection(0);
    }
  }
}


RT_PROGRAM void intersect( int primIdx )
{
    meshIntersect<false>( primIdx );
}


RT_PROGRAM void intersect_refine( int primIdx )
{
    meshIntersect<true>( primIdx );
}


RT_PROGRAM void bounds (int primIdx, float result[6])
{
  const int3 v_idx = indices_buffer[primIdx];

  const float3 v0   = vertices_buffer[ v_idx.x ];
  const float3 v1   = vertices_buffer[ v_idx.y ];
  const float3 v2   = vertices_buffer[ v_idx.z ];
  const float  area = length(cross(v1-v0, v2-v0));

  optix::Aabb* aabb = (optix::Aabb*)result;

  if(area > 0.0f && !isinf(area)) {
    aabb->m_min = fminf( fminf( v0, v1), v2 );
    aabb->m_max = fmaxf( fmaxf( v0, v1), v2 );
  } else {
    aabb->invalidate();
  }
}

