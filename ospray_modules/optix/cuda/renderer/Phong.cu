#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2016, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/*
 * Cyrille Favreau <cyrille.favreau@epfl.ch>
 * Added support for electron shading, shadows, soft shadows and ambient occlusion
 */

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include "Phong.h"

using namespace optix;

// Material attributes
rtDeclareVariable(float3,       Ka, , );
rtDeclareVariable(float3,       Kd, , );
rtDeclareVariable(float3,       Ks, , );
rtDeclareVariable(float3,       Kr, , );
rtDeclareVariable(float3,       Ko, , );
rtDeclareVariable(float,        glossiness, , );
rtDeclareVariable(unsigned int, shading_mode, , );
rtDeclareVariable(float,        refraction_index, , );
rtDeclareVariable(float,        phong_exp, , );

rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, ); 
rtDeclareVariable(float3, shading_normal, attribute shading_normal, ); 

// Textures
rtTextureSampler<float4, 2> diffuse_map;
rtDeclareVariable(float3, texcoord, attribute texcoord, );


RT_PROGRAM void any_hit_shadow()
{
    phongShadowed( Ko );
}


RT_PROGRAM void closest_hit_radiance()
{
    float3 world_shading_normal   = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );
    float3 world_geometric_normal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );

    float3 ffnormal = faceforward( world_shading_normal, -ray.direction, world_geometric_normal );
    phongShade( Kd, Ka, Ks, Kr, Ko, refraction_index, phong_exp, glossiness, shading_mode, ffnormal, ray.tmax );
}


RT_PROGRAM void closest_hit_radiance_textured()
{
    float3 world_shading_normal   = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );
    float3 world_geometric_normal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );

    float3 ffnormal = faceforward( world_shading_normal, -ray.direction, world_geometric_normal );

    const float3 Kd = make_float3( tex2D( diffuse_map, texcoord.x, texcoord.y ) );
    phongShade( Kd, Ka, Ks, Kr, Ko, refraction_index, phong_exp, glossiness, shading_mode, ffnormal, ray.tmax );
}
