#include "hip/hip_runtime.h"
/* Copyright (c) 2019, EPFL/Blue Brain Project
 * All rights reserved. Do not distribute without permission.
 *
 * This file is part of Brayns <https://github.com/BlueBrain/Brayns>
 *
 * This library is free software; you can redistribute it and/or modify it under
 * the terms of the GNU Lesser General Public License version 3.0 as published
 * by the Free Software Foundation.
 *
 * This library is distributed in the hope that it will be useful, but WITHOUT
 * ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS
 * FOR A PARTICULAR PURPOSE.  See the GNU Lesser General Public License for more
 * details.
 *
 * You should have received a copy of the GNU Lesser General Public License
 * along with this library; if not, write to the Free Software Foundation, Inc.,
 * 51 Franklin Street, Fifth Floor, Boston, MA 02110-1301 USA.
 */

#include "../../CommonStructs.h"
#include "../Helpers.h"
#include <optix_world.h>

using namespace optix;

// Scene
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(PerRayData_radiance, prd, rtPayload, );
rtDeclareVariable(float, t_hit, rtIntersectionDistance, );

// Material attributes

rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );

// Textures
rtDeclareVariable(int, albedoMetallic_map, , );
rtDeclareVariable(int, normalRoughness_map, , );
rtDeclareVariable(float2, texcoord, attribute texcoord, );

// Lights
rtBuffer<BasicLight> lights;

// Vertices
rtDeclareVariable(float3, v0, attribute v0, );
rtDeclareVariable(float3, v1, attribute v1, );
rtDeclareVariable(float3, v2, attribute v2, );
rtDeclareVariable(float2, t0, attribute t0, );
rtDeclareVariable(float2, t1, attribute t1, );
rtDeclareVariable(float2, t2, attribute t2, );

rtDeclareVariable(float2, ddx, attribute ddx, );
rtDeclareVariable(float2, ddy, attribute ddy, );

rtDeclareVariable(uint, use_envmap, , );
rtDeclareVariable(int, envmap_radiance, , );
rtDeclareVariable(int, envmap_irradiance, , );
rtDeclareVariable(int, envmap_brdf_lut, , );
rtDeclareVariable(uint, radianceLODs, , );

static __device__ inline float calculateAttenuation(float3 WorldPos,
                                                    float3 lightPos)
{
    float distance = length(lightPos - WorldPos);
    return 1.0f / (distance * distance);
}

static __device__ inline float distributionGGX(float3 N, float3 H,
                                               float roughness)
{
    const float a = roughness * roughness;
    const float a2 = a * a;
    const float NdotH = max(dot(N, H), 0.0f);
    const float NdotH2 = NdotH * NdotH;

    float denom = (NdotH2 * (a2 - 1.0f) + 1.0f);
    denom = M_PIf * denom * denom;

    return a2 / denom;
}

static __device__ inline float GeometrySchlickGGX(float NdotV, float roughness)
{
    const float r = roughness + 1.0f;
    const float k = (r * r) / 8.0f;

    float denom = NdotV * (1.0 - k) + k;

    return NdotV / denom;
}

static __device__ inline float geometrySmith(float3 N, float3 V, float3 L,
                                             float roughness)
{
    const float NdotV = max(dot(N, V), 0.0f);
    const float NdotL = max(dot(N, L), 0.0f);
    const float ggx2 = GeometrySchlickGGX(NdotV, roughness);
    const float ggx1 = GeometrySchlickGGX(NdotL, roughness);

    return ggx1 * ggx2;
}

static __device__ inline float3 fresnelSchlick(float cosTheta, float3 F0)
{
    return F0 + (make_float3(1.0f) - F0) * pow(1.0f - cosTheta, 5.0f);
}

static __device__ inline float3 fresnelSchlickRoughness(float cosTheta,
                                                        float3 F0,
                                                        float roughness)
{
    return F0 +
           (max(make_float3(1.0f - roughness), F0) - F0) *
               pow(1.0f - cosTheta, 5.0f);
}

static __device__ inline void shade()
{
    const float3 world_shading_normal =
        optix::normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));

    float3 N = world_shading_normal;

    const float3 edge1 = v1 - v0;
    const float3 edge2 = v2 - v0;
    const float2 deltaUV1 = t1 - t0;
    const float2 deltaUV2 = t2 - t0;

    float f = 1.0f / (deltaUV1.x * deltaUV2.y - deltaUV2.x * deltaUV1.y);

    float3 tangent;
    tangent.x = f * (deltaUV2.y * edge1.x - deltaUV1.y * edge2.x);
    tangent.y = f * (deltaUV2.y * edge1.y - deltaUV1.y * edge2.y);
    tangent.z = f * (deltaUV2.y * edge1.z - deltaUV1.y * edge2.z);
    tangent = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, tangent));

    tangent = normalize(tangent - dot(tangent, N) * N);

    float3 bitangent = cross(N, tangent);

    const float3 WorldPos = ray.origin + t_hit * ray.direction;
    const float3 V = -ray.direction;
    const float4 albedoMetallic =
        SRGBtoLinear(rtTex2DGrad<float4>(albedoMetallic_map, texcoord.x,
                                         texcoord.y, ddx, ddy));
    float3 albedo = make_float3(albedoMetallic);

    const float4 normalRoughness =
        rtTex2D<float4>(normalRoughness_map, texcoord.x, texcoord.y);
    const float roughness = max(normalRoughness.w, 0.05f);
    const float3 normal = make_float3(normalRoughness);
    optix::Matrix3x3 TBN;
    TBN.setCol(0, tangent);
    TBN.setCol(1, bitangent);
    TBN.setCol(2, N);

    N = normalize(TBN * (normal * 2.0f - 1.0f));

    const float3 F0 = lerp(make_float3(0.04f), albedo, albedoMetallic.w);

    float3 Lo = make_float3(0.0f);
    unsigned int num_lights = lights.size();
    for (int i = 0; i < num_lights; ++i)
    {
        // per-light radiance
        // const BasicLight& light = lights[i];
        BasicLight light = lights[i];
        light.pos = make_float3(0.5f, 1.0f, 1.5f);
        const float3 L = normalize(light.pos - WorldPos);
        const float3 H = normalize(V + L);
        const float attenuation = calculateAttenuation(WorldPos, light.pos);
        const float3 radiance =
            light.color * attenuation * 20.0f;

        // cook-torrance brdf
        const float NDF = distributionGGX(N, H, roughness);
        const float G = geometrySmith(N, V, L, roughness);
        const float3 F = fresnelSchlick(max(dot(H, V), 0.0f), F0);

        const float3 kD = (make_float3(1.0f) - F) * (1.0f - albedoMetallic.w);

        const float3 numerator = NDF * G * F;
        const float NdotL = max(dot(N, L), 0.0f);
        const float denominator = 4.0f * max(dot(N, V), 0.0f) * NdotL;
        const float3 specular = numerator / max(denominator, 0.001f);

        Lo += (kD * albedo / M_PIf + specular) * radiance * NdotL;
    }

    float3 ambient = make_float3(0.03f) * albedo /* * ao*/;
    if (use_envmap)
    {
        const float NdotV = dot(N, V);
        const float3 F =
            fresnelSchlickRoughness(max(NdotV, 0.0f), F0, roughness);
        const float3 kD = (make_float3(1.0f) - F) * (1.0f - albedoMetallic.w);

        const float3 irradiance = make_float3(
            rtTexCubemap<float4>(envmap_irradiance, N.x, N.y, N.z));
        const float3 diffuse = irradiance * albedo;

        // sample both the pre-filter map and the BRDF lut and combine them
        // together as per the Split-Sum approximation to get the IBL specular
        // part.
        const float3 reflectV = reflect(-V, N);
        const float3 prefilteredColor = make_float3(
            rtTexCubemapLod<float4>(envmap_radiance, reflectV.x, reflectV.y, reflectV.z,
                               roughness * float(radianceLODs)));
        const float2 brdf = make_float2(
            rtTex2D<float4>(envmap_brdf_lut, max(NdotV, 0.0), roughness));
        const float3 specular = prefilteredColor * (F * brdf.x + brdf.y);

        ambient = (kD * diffuse + specular) /* * ao*/;
    }

    const float3 color = ambient + Lo;
    prd.result = linearToSRGB(tonemap(color));
}

RT_PROGRAM void any_hit_shadow()
{
    rtTerminateRay();
}

RT_PROGRAM void closest_hit_radiance()
{
    shade();
}
