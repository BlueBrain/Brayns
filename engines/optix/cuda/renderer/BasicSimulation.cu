#include "hip/hip_runtime.h"
/* Copyright (c) 2019, EPFL/Blue Brain Project
 * All rights reserved. Do not distribute without permission.
 *
 * This file is part of Brayns <https://github.com/BlueBrain/Brayns>
 *
 * This library is free software; you can redistribute it and/or modify it under
 * the terms of the GNU Lesser General Public License version 3.0 as published
 * by the Free Software Foundation.
 *
 * This library is distributed in the hope that it will be useful, but WITHOUT
 * ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS
 * FOR A PARTICULAR PURPOSE.  See the GNU Lesser General Public License for more
 * details.
 *
 * You should have received a copy of the GNU Lesser General Public License
 * along with this library; if not, write to the Free Software Foundation, Inc.,
 * 51 Franklin Street, Fifth Floor, Boston, MA 02110-1301 USA.
 */

#include <optix_world.h>

struct PerRayData_radiance
{
    float3 result;
    float importance;
    int depth;
};

// Scene
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(PerRayData_radiance, prd, rtPayload, );

// Material attributes
rtDeclareVariable(float3, Kd, , );

rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );

// Textures
rtTextureSampler<float4, 2> diffuse_map;
rtDeclareVariable(float3, texcoord, attribute texcoord, );

// Simulation data
rtDeclareVariable(unsigned int, use_simulation_data, , );
rtBuffer<float3> colors;
rtBuffer<float> opacities;
rtDeclareVariable(float2, value_range, , );
rtBuffer<float> simulation_data;
rtDeclareVariable(unsigned long, user_data, attribute user_data, );

struct InterpolatedValue
{
    float influence_low;
    float influence_high;
    int idx_low;
    int idx_high;
};

// Finds between what pair of values a given value is and how much influence
// each of these values have
static __device__ inline InterpolatedValue interpolateValues(
    const float v_min, const float v_max, const float value,
    const int num_values)
{
    InterpolatedValue result;

    const float v_clamped = min(v_max, max(v_min, value));
    const float range_per_value = (v_max - v_min) / (num_values - 1);
    const float idx_value = (v_clamped - v_min) / range_per_value;

    // Indices of the two values to interpolate between
    const int idx_value_low = max(0, int(floor(idx_value)));
    const int idx_value_high = min(num_values - 1, int(ceil(idx_value)));

    result.idx_low = idx_value_low;
    result.idx_high = idx_value_high;

    if (idx_value_low == idx_value_high)
    {
        // Since both values are the same we split the influence
        result.influence_low = 0.5f;
        result.influence_high = 0.5f;
    }
    else
    {
        const float v_low = v_min + float(idx_value_low) * range_per_value;
        const float v_high = v_min + float(idx_value_high) * range_per_value;

        result.influence_low = 1.0f - abs(v_clamped - v_low) / range_per_value;
        result.influence_high =
            1.0f - abs(v_clamped - v_high) / range_per_value;
    }

    return result;
}

static __device__ inline float3 calcTransferFunctionColor()
{
    const float3 WHITE = make_float3(1.f, 1.f, 1.f);
    float3 color_opaque;
    float opacity;

    const float x_min = value_range.x;
    const float x_max = value_range.y;
    const float x_value = simulation_data[user_data];

    {
        const int num_colors = colors.size() / (sizeof(float3));
        const InterpolatedValue iv =
            interpolateValues(x_min, x_max, x_value, num_colors);
        color_opaque = colors[iv.idx_low] * iv.influence_low +
                       colors[iv.idx_high] * iv.influence_high;
    }

    {
        const int num_opacities = opacities.size() / (sizeof(float));
        const InterpolatedValue iv =
            interpolateValues(x_min, x_max, x_value, num_opacities);
        opacity = opacities[iv.idx_low] * iv.influence_low +
                  opacities[iv.idx_high] * iv.influence_high;
    }

    return opacity * color_opaque + (1.0f - opacity) * WHITE;
}

static __device__ inline void shade(bool textured)
{
    float3 world_shading_normal =
        optix::normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
    float3 world_geometric_normal = optix::normalize(
        rtTransformNormal(RT_OBJECT_TO_WORLD, geometric_normal));

    float3 p_normal = optix::faceforward(world_shading_normal, -ray.direction,
                                         world_geometric_normal);

    const float3 p_Kd =
        textured ? make_float3(tex2D(diffuse_map, texcoord.x, texcoord.y)) : Kd;

    prd.result = p_Kd * max(0.f, optix::dot(-ray.direction, p_normal));
    const float3 col = use_simulation_data ? calcTransferFunctionColor() : Kd;
    prd.result = col * max(0.f, optix::dot(-ray.direction, p_normal));
}

RT_PROGRAM void any_hit_shadow()
{
    rtTerminateRay();
}

RT_PROGRAM void closest_hit_radiance()
{
    shade(false);
}

RT_PROGRAM void closest_hit_radiance_textured()
{
    shade(true);
}
