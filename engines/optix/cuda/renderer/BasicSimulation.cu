#include "hip/hip_runtime.h"
/* Copyright (c) 2019, EPFL/Blue Brain Project
 * All rights reserved. Do not distribute without permission.
 *
 * This file is part of Brayns <https://github.com/BlueBrain/Brayns>
 *
 * This library is free software; you can redistribute it and/or modify it under
 * the terms of the GNU Lesser General Public License version 3.0 as published
 * by the Free Software Foundation.
 *
 * This library is distributed in the hope that it will be useful, but WITHOUT
 * ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS
 * FOR A PARTICULAR PURPOSE.  See the GNU Lesser General Public License for more
 * details.
 *
 * You should have received a copy of the GNU Lesser General Public License
 * along with this library; if not, write to the Free Software Foundation, Inc.,
 * 51 Franklin Street, Fifth Floor, Boston, MA 02110-1301 USA.
 */

#include <optix_world.h>

struct PerRayData_radiance
{
    float3 result;
    float importance;
    int depth;
};

// Scene
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(PerRayData_radiance, prd, rtPayload, );

// Material attributes
rtDeclareVariable(float3, Kd, , );

rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );

// Textures
rtTextureSampler<float4, 2> diffuse_map;
rtDeclareVariable(float3, texcoord, attribute texcoord, );

// Simulation data
rtDeclareVariable(unsigned int, use_simulation_data, , );
rtBuffer<float3> colors;
rtBuffer<float> opacities;
rtDeclareVariable(float2, value_range, , );
rtBuffer<float> simulation_data;
rtDeclareVariable(unsigned long, simulation_idx, attribute simulation_idx, );

template <typename T>
static __device__ inline T interpolateValues(const float v_min,
                                             const float v_max,
                                             const float value,
                                             optix::buffer<T, 1> &values)
{
    const int num_values = values.size() / sizeof(T);

    const float v_clamped = min(v_max, max(v_min, value));
    const float range_per_value = (v_max - v_min) / (num_values - 1);
    const float idx_value = (v_clamped - v_min) / range_per_value;

    const int index = int(floor(idx_value));

    if (index == num_values - 1)
        return values[index];

    const float v_low = v_min + float(index) * range_per_value;
    const float t = (v_clamped - v_low) / range_per_value;

    return values[index] * (1.0f - t) + values[index + 1] * t;
}

static __device__ inline float3 calcTransferFunctionColor()
{
    const float3 WHITE = make_float3(1.f, 1.f, 1.f);

    const float x_min = value_range.x;
    const float x_max = value_range.y;
    const float x_value = simulation_data[simulation_idx];

    const float3 color_opaque =
        interpolateValues<float3>(x_min, x_max, x_value, colors);

    const float opacity =
        interpolateValues<float>(x_min, x_max, x_value, opacities);

    return opacity * color_opaque + (1.0f - opacity) * WHITE;
}

static __device__ inline void shade(bool textured)
{
    float3 world_shading_normal =
        optix::normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
    float3 world_geometric_normal = optix::normalize(
        rtTransformNormal(RT_OBJECT_TO_WORLD, geometric_normal));

    float3 p_normal = optix::faceforward(world_shading_normal, -ray.direction,
                                         world_geometric_normal);

    const float3 p_Kd =
        textured ? make_float3(tex2D(diffuse_map, texcoord.x, texcoord.y)) : Kd;
    const float3 color =
        use_simulation_data ? calcTransferFunctionColor() : p_Kd;

    prd.result = color * max(0.f, optix::dot(-ray.direction, p_normal));
}

RT_PROGRAM void any_hit_shadow()
{
    rtTerminateRay();
}

RT_PROGRAM void closest_hit_radiance()
{
    shade(false);
}

RT_PROGRAM void closest_hit_radiance_textured()
{
    shade(true);
}
