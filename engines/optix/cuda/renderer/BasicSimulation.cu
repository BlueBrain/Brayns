#include "hip/hip_runtime.h"
/* Copyright (c) 2019, EPFL/Blue Brain Project
 * All rights reserved. Do not distribute without permission.
 *
 * This file is part of Brayns <https://github.com/BlueBrain/Brayns>
 *
 * This library is free software; you can redistribute it and/or modify it under
 * the terms of the GNU Lesser General Public License version 3.0 as published
 * by the Free Software Foundation.
 *
 * This library is distributed in the hope that it will be useful, but WITHOUT
 * ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS
 * FOR A PARTICULAR PURPOSE.  See the GNU Lesser General Public License for more
 * details.
 *
 * You should have received a copy of the GNU Lesser General Public License
 * along with this library; if not, write to the Free Software Foundation, Inc.,
 * 51 Franklin Street, Fifth Floor, Boston, MA 02110-1301 USA.
 */

#include "TransferFunction.h"

#include <optix_world.h>

struct PerRayData_radiance
{
    float3 result;
    float importance;
    int depth;
};

// Scene
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(PerRayData_radiance, prd, rtPayload, );

// Material attributes
rtDeclareVariable(float3, Kd, , );

rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );

// Textures
rtDeclareVariable(int, albedoMetallic_map, , );
rtDeclareVariable(float2, texcoord, attribute texcoord, );

// Simulation data
rtBuffer<float3> colors;
rtBuffer<float> opacities;
rtDeclareVariable(float2, value_range, , );
rtBuffer<float> simulation_data;
rtDeclareVariable(unsigned long, simulation_idx, attribute simulation_idx, );

static __device__ inline void shade(bool textured)
{
    float3 world_shading_normal =
        optix::normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
    float3 world_geometric_normal = optix::normalize(
        rtTransformNormal(RT_OBJECT_TO_WORLD, geometric_normal));

    float3 p_normal = optix::faceforward(world_shading_normal, -ray.direction,
                                         world_geometric_normal);

    float3 p_Kd;
    if (simulation_data.size() > 0)
        p_Kd = calcTransferFunctionColor(value_range.x, value_range.y,
                                         simulation_data[simulation_idx],
                                         colors, opacities);
    else if (textured)
        p_Kd = make_float3(
            optix::rtTex2D<float4>(albedoMetallic_map, texcoord.x, texcoord.y));
    else
        p_Kd = Kd;

    prd.result = p_Kd * max(0.f, optix::dot(-ray.direction, p_normal));
}

RT_PROGRAM void any_hit_shadow()
{
    rtTerminateRay();
}

RT_PROGRAM void closest_hit_radiance()
{
    shade(false);
}

RT_PROGRAM void closest_hit_radiance_textured()
{
    shade(true);
}
