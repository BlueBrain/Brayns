#include "hip/hip_runtime.h"
/* Copyright (c) 2019, EPFL/Blue Brain Project
 * All rights reserved. Do not distribute without permission.
 *
 * This file is part of Brayns <https://github.com/BlueBrain/Brayns>
 *
 * This library is free software; you can redistribute it and/or modify it under
 * the terms of the GNU Lesser General Public License version 3.0 as published
 * by the Free Software Foundation.
 *
 * This library is distributed in the hope that it will be useful, but WITHOUT
 * ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS
 * FOR A PARTICULAR PURPOSE.  See the GNU Lesser General Public License for more
 * details.
 *
 * You should have received a copy of the GNU Lesser General Public License
 * along with this library; if not, write to the Free Software Foundation, Inc.,
 * 51 Franklin Street, Fifth Floor, Boston, MA 02110-1301 USA.
 */

// clang-format off

#include "../../CommonStructs.h"
#include "../Helpers.h"
#include <optix_world.h>

using namespace optix;

// Scene
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(PerRayData_radiance, prd, rtPayload, );
rtDeclareVariable(float, t_hit, rtIntersectionDistance, );
rtDeclareVariable(float, currentTime, , );

// Material attributes

rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );

// Textures
rtDeclareVariable(int, albedoMetallic_map, , );
rtDeclareVariable(int, normalRoughness_map, , );
rtDeclareVariable(int, aoEmissive_map, , );
rtDeclareVariable(float2, texcoord, attribute texcoord, );

// Lights
rtBuffer<BasicLight> lights;

// Vertices
rtDeclareVariable(float3, v0, attribute v0, );
rtDeclareVariable(float3, v1, attribute v1, );
rtDeclareVariable(float3, v2, attribute v2, );
rtDeclareVariable(float2, t0, attribute t0, );
rtDeclareVariable(float2, t1, attribute t1, );
rtDeclareVariable(float2, t2, attribute t2, );

rtDeclareVariable(float3, ddxWPos, attribute ddxWPos, );
rtDeclareVariable(float3, ddyWPos, attribute ddyWPos, );

rtDeclareVariable(uint, use_envmap, , );
rtDeclareVariable(int, envmap_radiance, , );
rtDeclareVariable(int, envmap_irradiance, , );
rtDeclareVariable(int, envmap_brdf_lut, , );
rtDeclareVariable(uint, radianceLODs, , );

static __device__ inline optix::Matrix3x3 cotangentFrame(float3 N, float3 ddxwpos, float3 ddywpos, float2 ddxuv, float2 ddyuv)
{
     // solve the linear system
    float3 dp2perp = cross( ddywpos, N );
    float3 dp1perp = cross( N, ddxwpos );
    float3 T = dp2perp * ddxuv.x + dp1perp * ddyuv.x;
    float3 B = dp2perp * ddxuv.y + dp1perp * ddyuv.y;

    // construct a scale-invariant frame
    float invmax = rsqrt(max(dot(T,T), dot(B,B)));

    optix::Matrix3x3 TBN;
    TBN.setCol(0, T * invmax);
    TBN.setCol(1, B * invmax);
    TBN.setCol(2, N);
    return TBN;
}

static __device__ inline float calculateAttenuation(float3 WorldPos,
                                                    float3 lightPos)
{
    float distance = length(lightPos - WorldPos);
    return 1.0f / (distance * distance);
}

static __device__ inline float distributionGGX(float3 N, float3 H,
                                               float roughness)
{
    const float a = roughness * roughness;
    const float a2 = a * a;
    const float NdotH = max(dot(N, H), 0.0f);
    const float NdotH2 = NdotH * NdotH;

    float denom = (NdotH2 * (a2 - 1.0f) + 1.0f);
    denom = M_PIf * denom * denom;

    return a2 / denom;
}

static __device__ inline float GeometrySchlickGGX(float NdotV, float roughness)
{
    const float r = roughness + 1.0f;
    const float k = (r * r) / 8.0f;

    float denom = NdotV * (1.0 - k) + k;

    return NdotV / denom;
}

static __device__ inline float geometrySmith(float3 N, float3 V, float3 L,
                                             float roughness)
{
    const float NdotV = max(dot(N, V), 0.0f);
    const float NdotL = max(dot(N, L), 0.0f);
    const float ggx2 = GeometrySchlickGGX(NdotV, roughness);
    const float ggx1 = GeometrySchlickGGX(NdotL, roughness);

    return ggx1 * ggx2;
}

static __device__ inline float3 fresnelSchlick(float cosTheta, float3 F0)
{
    return F0 + (make_float3(1.0f) - F0) * pow(1.0f - cosTheta, 5.0f);
}

static __device__ inline float3 fresnelSchlickRoughness(float cosTheta,
                                                        float3 F0,
                                                        float roughness)
{
    return F0 +
           (max(make_float3(1.0f - roughness), F0) - F0) *
               pow(1.0f - cosTheta, 5.0f);
}

static __device__ inline void shade()
{
    const float3 world_shading_normal =
        optix::normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal)); 

    float3 N = world_shading_normal;

    const float3 WorldPos = ray.origin + t_hit * ray.direction;
    const float3 V = -ray.direction;

    // ******* triplanar ********
    float3 triblend = N * N;
    triblend = triblend / (triblend.x + triblend.y + triblend.z);

    const float planeSizeRatio = /*0.11f*/ 12.0f; // must be parameter
    const float3 uv = planeSizeRatio * WorldPos;
    const float3 ddxuv = planeSizeRatio * ddxWPos;
    const float3 ddyuv = planeSizeRatio * ddyWPos; 

    const float2 ddxuvX = make_float2(ddxuv.z, ddxuv.y);
    const float2 ddxuvY = make_float2(ddxuv.x, ddxuv.z);
    const float2 ddxuvZ = make_float2(ddxuv.x, ddxuv.y);

    const float2 ddyuvX = make_float2(ddyuv.z, ddyuv.y);
    const float2 ddyuvY = make_float2(ddyuv.x, ddyuv.z);
    const float2 ddyuvZ = make_float2(ddyuv.x, ddyuv.y); 

    const float4 albedoMetallicX = SRGBtoLinear(rtTex2DGrad<float4>(albedoMetallic_map, uv.z, uv.y, ddxuvX, ddyuvX)); 
    const float4 albedoMetallicY = SRGBtoLinear(rtTex2DGrad<float4>(albedoMetallic_map, uv.x, uv.z, ddxuvY, ddyuvY));
    const float4 albedoMetallicZ = SRGBtoLinear(rtTex2DGrad<float4>(albedoMetallic_map, uv.x, uv.y, ddxuvZ, ddyuvZ));
    const float4 albedoMetallic = triblend.y * albedoMetallicY + triblend.x * albedoMetallicX + triblend.z * albedoMetallicZ;
    const float3 albedo = make_float3(albedoMetallic);

    const float4 aoEmissiveX = rtTex2DGrad<float4>(aoEmissive_map, uv.z, uv.y, ddxuvX, ddyuvX);
    const float4 aoEmissiveY = rtTex2DGrad<float4>(aoEmissive_map, uv.x, uv.z, ddxuvY, ddyuvY);
    const float4 aoEmissiveZ = rtTex2DGrad<float4>(aoEmissive_map, uv.x, uv.y, ddxuvZ, ddyuvZ);
    const float4 aoEmissive = triblend.y * aoEmissiveY + triblend.x * aoEmissiveX + triblend.z * aoEmissiveZ;
    const float ao = aoEmissive.x;
    const float emissive = aoEmissive.y;    

    const float4 normalRoughnessX = rtTex2DGrad<float4>(normalRoughness_map, uv.z, uv.y, ddxuvX, ddyuvX );
    const float4 normalRoughnessY = rtTex2DGrad<float4>(normalRoughness_map, uv.x, uv.z, ddxuvY, ddyuvY );
    const float4 normalRoughnessZ = rtTex2DGrad<float4>(normalRoughness_map, uv.x, uv.y, ddxuvZ, ddyuvZ );

    float3 normalX = 2.0f * make_float3(normalRoughnessX) - 1.0f;
    float3 normalY = 2.0f * make_float3(normalRoughnessY) - 1.0f;
    float3 normalZ = 2.0f * make_float3(normalRoughnessZ) - 1.0f;

    /*** TANGENT SPACE MATRIX TECHNIQUE ***/

    // Calculate the cotangent frame for each plane
    const optix::Matrix3x3 tbnX = cotangentFrame(N, ddxWPos, ddyWPos, ddxuvX, ddyuvX );
    const optix::Matrix3x3 tbnY = cotangentFrame(N, ddxWPos, ddyWPos, ddxuvY, ddyuvY );
    const optix::Matrix3x3 tbnZ = cotangentFrame(N, ddxWPos, ddyWPos, ddxuvZ, ddyuvZ );

    // Apply cotangent frame and triblend normals
    N = normalize(tbnX * normalX * triblend.x + tbnY * normalY * triblend.y + tbnZ * normalZ * triblend.z);
   
    /*** TANGENT SPACE MATRIX TECHNIQUE ***/

    const float roughnessX = normalRoughnessX.w;
    const float roughnessY = normalRoughnessY.w;
    const float roughnessZ = normalRoughnessZ.w;
    const float roughness = triblend.y * roughnessY + triblend.x * roughnessX + triblend.z * roughnessZ;

    const float3 F0 = lerp(make_float3(0.04f), albedo, albedoMetallic.w);

    float3 Lo = make_float3(0.0f);
    unsigned int num_lights =  lights.size();
    for (int i = 0; i < num_lights; ++i)
    {
        // per-light radiance
        BasicLight light = lights[i];
        light.pos = make_float3(5.0f, 10.0f, 15.0f);
        const float3 L = normalize(light.pos - WorldPos);
        const float3 H = normalize(V + L);
        const float attenuation = calculateAttenuation(WorldPos, light.pos);
        const float3 radiance =
            light.color * attenuation * 45.0f;

        // cook-torrance brdf
        const float NDF = distributionGGX(N, H, roughness);
        const float G = geometrySmith(N, V, L, roughness);
        const float3 F = fresnelSchlick(max(dot(H, V), 0.0f), F0);

        const float3 kD = (make_float3(1.0f) - F) * (1.0f - albedoMetallic.w);

        const float3 numerator = NDF * G * F;
        const float NdotL = max(dot(N, L), 0.0f);
        const float denominator = 4.0f * max(dot(N, V), 0.0f) * NdotL;
        const float3 specular = numerator / max(denominator, 0.001f);

        Lo += (kD * albedo / M_PIf + specular) * radiance * NdotL;
    }

    float3 ambient = make_float3(0.005f) * albedo * ao;
    if (use_envmap)
    {
        const float NdotV = dot(N, V);
        const float3 F =
            fresnelSchlickRoughness(max(NdotV, 0.0f), F0, roughness);
        const float3 kD = (make_float3(1.0f) - F) * (1.0f - albedoMetallic.w);

        const float3 irradiance = make_float3(
            rtTexCubemap<float4>(envmap_irradiance, N.x, N.y, N.z));
        const float3 diffuse = irradiance * albedo;

        // sample both the pre-filter map and the BRDF lut and combine them
        // together as per the Split-Sum approximation to get the IBL specular
        // part.
        const float3 reflectV = reflect(-V, N);
        const float3 prefilteredColor = make_float3(
            rtTexCubemapLod<float4>(envmap_radiance, reflectV.x, reflectV.y, reflectV.z,
                               roughness * float(radianceLODs)));
        const float2 brdf = make_float2(
            rtTex2D<float4>(envmap_brdf_lut, max(NdotV, 0.0), roughness));
        const float3 specular = prefilteredColor * (F * brdf.x + brdf.y);

        ambient = (kD * diffuse + specular) * ao;
    }
    const int size = 28;
    float3 colormap[size] = {make_float3(0.0f, 0.0f, 1.0f), make_float3(0.0f, 0.0f, 1.0f), make_float3(0.0f, 0.0f, 1.0f), make_float3(0.0f, 0.0f, 1.0f),
                             make_float3(0.0f, 0.0f, 1.0f), make_float3(0.0f, 0.0f, 1.0f), make_float3(0.0f, 0.0f, 1.0f), make_float3(0.0f, 0.0f, 1.0f),
                             make_float3(0.0f, 0.0f, 1.0f), make_float3(0.0f, 0.0f, 1.0f), make_float3(0.0f, 0.0f, 1.0f), make_float3(0.0f, 0.0f, 1.0f),
                             make_float3(0.0f, 0.0f, 1.0f), make_float3(0.1f, 0.0f, 1.0f), make_float3(0.2f, 0.0f, 1.0f), make_float3(0.4f, 0.0f, 1.0f),
                             make_float3(0.75f, 0.0f, 1.0f), make_float3(1.0f, 0.0f, 0.9f), make_float3(1.0f, 0.0f, 0.8f), make_float3(1.0f, 0.0f, 0.6f),
                             make_float3(1.0f, 0.0f, 0.4f), make_float3(1.0f, 0.0f, 0.2f), make_float3(1.0f, 0.0f, 0.2f), make_float3(1.0f, 0.0f, 0.0f),
                             make_float3(1.0f, 0.0f, 0.0f), make_float3(1.0f, 0.0f, 0.0f), make_float3(1.0f, 0.0f, 0.0f), make_float3(1.0f, 0.0f, 0.0f)};
    float powermap[size] = {1.0f, 1.0f, 1.0f, 1.0f,
                            1.0f, 1.0f, 1.0f, 1.0f,
                            1.0f, 1.0f, 1.0f, 1.0f,
                            1.0f, 1.0f, 1.0f, 1.0f,
                            1.125f, 1.25f, 1.5f, 2.0f,
                            3.0f, 5.0f, 8.0f, 13.0f, 
                            21.0f, 34.0f, 55.0f, 89.0f};
    float period = 15.0f;
    float loopedTime = fmod(currentTime, period);
    float indexfloat = (size - 1) * loopedTime / period;
    int index = indexfloat;
    float frac = indexfloat - index;
    float3 emmissiveColor = ((1.0f - frac) * colormap[index] + frac * colormap[index + 1]) * ((1.0f - frac) * powermap[index] + frac * powermap[index + 1]);
    

    const float3 color = ambient + Lo + emissive * emmissiveColor;
    prd.result = linearToSRGB(tonemap(color));
}

RT_PROGRAM void any_hit_shadow()
{
    rtTerminateRay();
}

RT_PROGRAM void closest_hit_radiance()
{
    shade();
}

// clang-format on
