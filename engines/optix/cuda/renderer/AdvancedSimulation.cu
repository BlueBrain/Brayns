#include "hip/hip_runtime.h"
/* Copyright (c) 2019, EPFL/Blue Brain Project
 * All rights reserved. Do not distribute without permission.
 *
 * This file is part of Brayns <https://github.com/BlueBrain/Brayns>
 *
 * This library is free software; you can redistribute it and/or modify it under
 * the terms of the GNU Lesser General Public License version 3.0 as published
 * by the Free Software Foundation.
 *
 * This library is distributed in the hope that it will be useful, but WITHOUT
 * ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS
 * FOR A PARTICULAR PURPOSE.  See the GNU Lesser General Public License for more
 * details.
 *
 * You should have received a copy of the GNU Lesser General Public License
 * along with this library; if not, write to the Free Software Foundation, Inc.,
 * 51 Franklin Street, Fifth Floor, Boston, MA 02110-1301 USA.
 */

#include "AdvancedSimulation.h"
#include "TransferFunction.h"

#include <optix.h>
#include <optixu/optixu_math_namespace.h>

using namespace optix;

// Material attributes
rtDeclareVariable(float3, Ka, , );
rtDeclareVariable(float3, Kd, , );
rtDeclareVariable(float3, Ks, , );
rtDeclareVariable(float3, Kr, , );
rtDeclareVariable(float3, Ko, , );
rtDeclareVariable(float, glossiness, , );
rtDeclareVariable(float, refraction_index, , );
rtDeclareVariable(float, phong_exp, , );

rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );

// Textures
rtTextureSampler<float4, 2> diffuse_map;
rtDeclareVariable(float3, texcoord, attribute texcoord, );

// Simulation data
rtBuffer<float3> colors;
rtBuffer<float> opacities;
rtDeclareVariable(float2, value_range, , );
rtBuffer<float> simulation_data;
rtDeclareVariable(unsigned long, simulation_idx, attribute simulation_idx, );

RT_PROGRAM void any_hit_shadow()
{
    phongShadowed(Ko);
}

static __device__ inline void shade(bool textured)
{
    float3 world_shading_normal =
        normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
    float3 world_geometric_normal =
        normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, geometric_normal));

    float3 ffnormal = faceforward(world_shading_normal, -ray.direction,
                                  world_geometric_normal);

    float3 p_Kd;
    if (simulation_data.size() > 0)
        p_Kd = calcTransferFunctionColor(value_range.x, value_range.y,
                                         simulation_data[simulation_idx],
                                         colors, opacities);
    else if (textured)
        p_Kd = make_float3(tex2D(diffuse_map, texcoord.x, texcoord.y));
    else
        p_Kd = Kd;

    phongShade(p_Kd, Ka, Ks, Kr, Ko, refraction_index, phong_exp, glossiness,
               ffnormal, ray.tmax);
}

RT_PROGRAM void closest_hit_radiance()
{
    shade(false);
}

RT_PROGRAM void closest_hit_radiance_textured()
{
    shade(true);
}
