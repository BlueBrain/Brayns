#include "hip/hip_runtime.h"
/* Copyright (c) 2019, EPFL/Blue Brain Project
 * All rights reserved. Do not distribute without permission.
 *
 * This file is part of Brayns <https://github.com/BlueBrain/Brayns>
 *
 * This library is free software; you can redistribute it and/or modify it under
 * the terms of the GNU Lesser General Public License version 3.0 as published
 * by the Free Software Foundation.
 *
 * This library is distributed in the hope that it will be useful, but WITHOUT
 * ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS
 * FOR A PARTICULAR PURPOSE.  See the GNU Lesser General Public License for more
 * details.
 *
 * You should have received a copy of the GNU Lesser General Public License
 * along with this library; if not, write to the Free Software Foundation, Inc.,
 * 51 Franklin Street, Fifth Floor, Boston, MA 02110-1301 USA.
 */

#include "../../CommonStructs.h"
#include "IntersectionRefinement.h"
#include <optix.h>
#include <optixu/optixu_aabb_namespace.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_matrix_namespace.h>

using namespace optix;

// This is to be plugged into an RTgeometry object to represent
// a triangle mesh with a vertex buffer of triangle soup (triangle list)
// with an interleaved position, normal, texturecoordinate layout.

rtBuffer<float3> vertices_buffer;
rtBuffer<float3> normal_buffer;
rtBuffer<float2> texcoord_buffer;
rtBuffer<int3> indices_buffer;

rtDeclareVariable(float2, texcoord, attribute texcoord, );
rtDeclareVariable(float3, v0, attribute v0, );
rtDeclareVariable(float3, v1, attribute v1, );
rtDeclareVariable(float3, v2, attribute v2, );
rtDeclareVariable(float2, t0, attribute t0, );
rtDeclareVariable(float2, t1, attribute t1, );
rtDeclareVariable(float2, t2, attribute t2, );
rtDeclareVariable(float2, ddx, attribute ddx, );
rtDeclareVariable(float2, ddy, attribute ddy, );
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );

rtDeclareVariable(float3, back_hit_point, attribute back_hit_point, );
rtDeclareVariable(float3, front_hit_point, attribute front_hit_point, );

rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(PerRayData_radiance, prd, rtPayload, );
rtDeclareVariable(unsigned long, simulation_idx, attribute simulation_idx, );

static __device__ bool intersect_triangle_filtered(
    const Ray& ray, const float3& p0, const float3& p1, const float3& p2,
    const float3& rayDdx, const float3& rayDdy, float3& n, float& t,
    float& beta, float& gamma, float2& betaDerivative, float2& gammaDerivative)
{
    const float3 e0 = p1 - p0;
    const float3 e1 = p0 - p2;
    n = cross(e1, e0);

    const float3 g0 = p0 - ray.origin;

    const float NdotRay = dot(n, ray.direction);

    float3 e2 = g0 / NdotRay;
    float3 i = cross(ray.direction, e2);

    beta = dot(i, e1);
    gamma = dot(i, e0);
    t = dot(n, e2);

    if (!((t < ray.tmax) & (t > ray.tmin) & (beta >= 0.0f) & (gamma >= 0.0f) &
          (beta + gamma <= 1)))
        return false;

    const float3 differentialX =
        cross(rayDdx, e2) +
        cross(ray.direction, -e2 * dot(rayDdx, n) / NdotRay);
    const float3 differentialY =
        cross(rayDdy, e2) +
        cross(ray.direction, -e2 * dot(rayDdy, n) / NdotRay);
    betaDerivative.x = dot(differentialX, e1);
    betaDerivative.y = dot(differentialY, e1);
    gammaDerivative.x = dot(differentialX, e0);
    gammaDerivative.y = dot(differentialY, e0);

    return true;
}

template <bool DO_REFINE>
static __device__ void meshIntersect(int primIdx)
{
    const int3 v_idx = indices_buffer[primIdx];

    const float3 p0 = vertices_buffer[v_idx.x];
    const float3 p1 = vertices_buffer[v_idx.y];
    const float3 p2 = vertices_buffer[v_idx.z];

    // Intersect ray with triangle
    float3 n;
    float t;
    float beta, gamma;
    float2 betaDerivative, gammaDerivative;
    if (intersect_triangle_filtered(ray, p0, p1, p2, prd.rayDdx, prd.rayDdy, n,
                                    t, beta, gamma, betaDerivative,
                                    gammaDerivative))
    {
        if (rtPotentialIntersection(t))
        {
            v0 = p0;
            v1 = p1;
            v2 = p2;
            geometric_normal = normalize(n);
            if (normal_buffer.size() == 0)
                shading_normal = geometric_normal;
            else
            {
                float3 n0 = normal_buffer[v_idx.x];
                float3 n1 = normal_buffer[v_idx.y];
                float3 n2 = normal_buffer[v_idx.z];
                shading_normal = normalize(n1 * beta + n2 * gamma +
                                           n0 * (1.f - beta - gamma));
            }

            if (texcoord_buffer.size() == 0)
            {
                texcoord = make_float2(0.f, 0.f);
            }
            else
            {
                t0 = texcoord_buffer[v_idx.x];
                t1 = texcoord_buffer[v_idx.y];
                t2 = texcoord_buffer[v_idx.z];

                texcoord = t1 * beta + t2 * gamma + t0 * (1.f - beta - gamma);

                ddx = t1 * betaDerivative.x + t2 * gammaDerivative.x +
                      t0 * (-betaDerivative.x - gammaDerivative.x);
                ddy = t1 * betaDerivative.y + t2 * gammaDerivative.y +
                      t0 * (-betaDerivative.y - gammaDerivative.y);
            }

            if (DO_REFINE)
                refine_and_offset_hitpoint(ray.origin + t * ray.direction,
                                           ray.direction, geometric_normal, p0,
                                           back_hit_point, front_hit_point);
            simulation_idx = 0;
            rtReportIntersection(0);
        }
    }
}

RT_PROGRAM void intersect(int primIdx)
{
    meshIntersect<false>(primIdx);
}

RT_PROGRAM void intersect_refine(int primIdx)
{
    meshIntersect<true>(primIdx);
}

RT_PROGRAM void bounds(int primIdx, float result[6])
{
    const int3 v_idx = indices_buffer[primIdx];

    const float3 v0 = vertices_buffer[v_idx.x];
    const float3 v1 = vertices_buffer[v_idx.y];
    const float3 v2 = vertices_buffer[v_idx.z];
    const float area = length(cross(v1 - v0, v2 - v0));

    optix::Aabb* aabb = (optix::Aabb*)result;

    if (area > 0.0f && !isinf(area))
    {
        aabb->m_min = fminf(fminf(v0, v1), v2);
        aabb->m_max = fmaxf(fmaxf(v0, v1), v2);
    }
    else
    {
        aabb->invalidate();
    }
}
