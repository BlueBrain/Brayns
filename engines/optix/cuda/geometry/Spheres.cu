#include "hip/hip_runtime.h"
/* Copyright (c) 2019, EPFL/Blue Brain Project
 * All rights reserved. Do not distribute without permission.
 *
 * This file is part of Brayns <https://github.com/BlueBrain/Brayns>
 *
 * This library is free software; you can redistribute it and/or modify it under
 * the terms of the GNU Lesser General Public License version 3.0 as published
 * by the Free Software Foundation.
 *
 * This library is distributed in the hope that it will be useful, but WITHOUT
 * ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS
 * FOR A PARTICULAR PURPOSE.  See the GNU Lesser General Public License for more
 * details.
 *
 * You should have received a copy of the GNU Lesser General Public License
 * along with this library; if not, write to the Free Software Foundation, Inc.,
 * 51 Franklin Street, Fifth Floor, Boston, MA 02110-1301 USA.
 */

#include <optix_world.h>

using namespace optix;

#define OFFSET_USER_DATA 0
#define OFFSET_CENTER (OFFSET_USER_DATA + 2)
#define OFFSET_RADIUS (OFFSET_CENTER + 3)
#define OFFSET_TIMESTAMP (OFFSET_RADIUS + 1)
#define OFFSET_TEX_COORDS (OFFSET_TIMESTAMP + 1)

rtBuffer<float> spheres;

rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(unsigned int, sphere_size, , );

template <bool use_robust_method>
static __device__ void intersect_sphere(int primIdx)
{
    const int idx = primIdx * sphere_size;

    const float3 center = {spheres[idx + OFFSET_CENTER],
                           spheres[idx + OFFSET_CENTER + 1],
                           spheres[idx + OFFSET_CENTER + 2]};
    const float3 O = ray.origin - center;
    const float3 D = ray.direction;
    const float radius = spheres[idx + OFFSET_RADIUS];

    float b = dot(O, D);
    float c = dot(O, O) - radius * radius;
    float disc = b * b - c;
    if (disc > 0.0f)
    {
        float sdisc = sqrtf(disc);
        float root1 = (-b - sdisc);

        bool do_refine = false;

        float root11 = 0.0f;

        if (use_robust_method && fabsf(root1) > 10.f * radius)
            do_refine = true;

        if (do_refine)
        {
            // refine root1
            float3 O1 = O + root1 * ray.direction;
            b = dot(O1, D);
            c = dot(O1, O1) - radius * radius;
            disc = b * b - c;

            if (disc > 0.0f)
            {
                sdisc = sqrtf(disc);
                root11 = (-b - sdisc);
            }
        }

        bool check_second = true;
        if (rtPotentialIntersection(root1 + root11))
        {
            shading_normal = geometric_normal =
                (O + (root1 + root11) * D) / radius;
            if (rtReportIntersection(0))
                check_second = false;
        }
        if (check_second)
        {
            float root2 = (-b + sdisc) + (do_refine ? root1 : 0);
            if (rtPotentialIntersection(root2))
            {
                shading_normal = geometric_normal = (O + root2 * D) / radius;
                rtReportIntersection(0);
            }
        }
    }
}

RT_PROGRAM void intersect(int primIdx)
{
    intersect_sphere<false>(primIdx);
}

RT_PROGRAM void robust_intersect(int primIdx)
{
    intersect_sphere<true>(primIdx);
}

RT_PROGRAM void bounds(int primIdx, float result[6])
{
    const int idx = primIdx * sphere_size;
    const float3 cen = {spheres[idx + OFFSET_CENTER],
                        spheres[idx + OFFSET_CENTER + 1],
                        spheres[idx + OFFSET_CENTER + 2]};
    const float rad = spheres[idx + OFFSET_RADIUS];

    optix::Aabb* aabb = (optix::Aabb*)result;

    if (rad > 0.0f && !isinf(rad))
    {
        aabb->m_min = cen - rad;
        aabb->m_max = cen + rad;
    }
    else
    {
        aabb->invalidate();
    }
}
