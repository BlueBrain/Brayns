#include "hip/hip_runtime.h"
/* Copyright (c) 2019, EPFL/Blue Brain Project
 * All rights reserved. Do not distribute without permission.
 *
 * This file is part of Brayns <https://github.com/BlueBrain/Brayns>
 *
 * This library is free software; you can redistribute it and/or modify it under
 * the terms of the GNU Lesser General Public License version 3.0 as published
 * by the Free Software Foundation.
 *
 * This library is distributed in the hope that it will be useful, but WITHOUT
 * ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS
 * FOR A PARTICULAR PURPOSE.  See the GNU Lesser General Public License for more
 * details.
 *
 * You should have received a copy of the GNU Lesser General Public License
 * along with this library; if not, write to the Free Software Foundation, Inc.,
 * 51 Franklin Street, Fifth Floor, Boston, MA 02110-1301 USA.
 */

#include "IntersectionRefinement.h"
#include <optix.h>
#include <optixu/optixu_aabb_namespace.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_matrix_namespace.h>

using namespace optix;

// This is to be plugged into an RTgeometry object to represent
// a triangle mesh with a vertex buffer of triangle soup (triangle list)
// with an interleaved position, normal, texturecoordinate layout.

rtBuffer<float3> vertices_buffer;
rtBuffer<float3> normal_buffer;
rtBuffer<float2> texcoord_buffer;
rtBuffer<int3> indices_buffer;

rtDeclareVariable(float3, texcoord, attribute texcoord, );
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );

rtDeclareVariable(float3, back_hit_point, attribute back_hit_point, );
rtDeclareVariable(float3, front_hit_point, attribute front_hit_point, );

rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(unsigned long, simulation_idx, attribute simulation_idx, );

template <bool DO_REFINE>
static __device__ void meshIntersect(int primIdx)
{
    const int3 v_idx = indices_buffer[primIdx];

    const float3 p0 = vertices_buffer[v_idx.x];
    const float3 p1 = vertices_buffer[v_idx.y];
    const float3 p2 = vertices_buffer[v_idx.z];

    // Intersect ray with triangle
    float3 n;
    float t, beta, gamma;
    if (intersect_triangle(ray, p0, p1, p2, n, t, beta, gamma))
    {
        if (rtPotentialIntersection(t))
        {
            geometric_normal = normalize(n);
            if (normal_buffer.size() == 0)
                shading_normal = geometric_normal;
            else
            {
                float3 n0 = normal_buffer[v_idx.x];
                float3 n1 = normal_buffer[v_idx.y];
                float3 n2 = normal_buffer[v_idx.z];
                shading_normal = normalize(n1 * beta + n2 * gamma +
                                           n0 * (1.f - beta - gamma));
            }

            if (texcoord_buffer.size() == 0)
                texcoord = make_float3(0.f, 0.f, 0.f);
            else
            {
                float2 t0 = texcoord_buffer[v_idx.x];
                float2 t1 = texcoord_buffer[v_idx.y];
                float2 t2 = texcoord_buffer[v_idx.z];
                texcoord = make_float3(t1 * beta + t2 * gamma +
                                       t0 * (1.f - beta - gamma));
            }

            if (DO_REFINE)
                refine_and_offset_hitpoint(ray.origin + t * ray.direction,
                                           ray.direction, geometric_normal, p0,
                                           back_hit_point, front_hit_point);
            simulation_idx = 0;
            rtReportIntersection(0);
        }
    }
}

RT_PROGRAM void intersect(int primIdx)
{
    meshIntersect<false>(primIdx);
}

RT_PROGRAM void intersect_refine(int primIdx)
{
    meshIntersect<true>(primIdx);
}

RT_PROGRAM void bounds(int primIdx, float result[6])
{
    const int3 v_idx = indices_buffer[primIdx];

    const float3 v0 = vertices_buffer[v_idx.x];
    const float3 v1 = vertices_buffer[v_idx.y];
    const float3 v2 = vertices_buffer[v_idx.z];
    const float area = length(cross(v1 - v0, v2 - v0));

    optix::Aabb* aabb = (optix::Aabb*)result;

    if (area > 0.0f && !isinf(area))
    {
        aabb->m_min = fminf(fminf(v0, v1), v2);
        aabb->m_max = fmaxf(fmaxf(v0, v1), v2);
    }
    else
    {
        aabb->invalidate();
    }
}
