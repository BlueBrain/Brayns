#include "hip/hip_runtime.h"
/* Copyright (c) 2019, EPFL/Blue Brain Project
 * All rights reserved. Do not distribute without permission.
 *
 * This file is part of Brayns <https://github.com/BlueBrain/Brayns>
 *
 * This library is free software; you can redistribute it and/or modify it under
 * the terms of the GNU Lesser General Public License version 3.0 as published
 * by the Free Software Foundation.
 *
 * This library is distributed in the hope that it will be useful, but WITHOUT
 * ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS
 * FOR A PARTICULAR PURPOSE.  See the GNU Lesser General Public License for more
 * details.
 *
 * You should have received a copy of the GNU Lesser General Public License
 * along with this library; if not, write to the Free Software Foundation, Inc.,
 * 51 Franklin Street, Fifth Floor, Boston, MA 02110-1301 USA.
 */

#include <optix_world.h>

#include "Helpers.h"

struct PerRayData_radiance
{
    float3 result;
    float importance;
    int depth;
};

rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(float3, bgColor, , );
rtDeclareVariable(int, envmap, , );
rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );
rtDeclareVariable(uint, use_envmap, , );

RT_PROGRAM void envmap_miss()
{
    if (use_envmap)
    {
        const float2 uv = getEquirectangularUV(ray.direction);
        prd_radiance.result = linearToSRGB(
            tonemap(make_float3(optix::rtTex2D<float4>(envmap, uv.x, uv.y))));
    }
    else
    {
        prd_radiance.result = bgColor;
    }
}
