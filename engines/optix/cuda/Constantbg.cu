#include "hip/hip_runtime.h"
/* Copyright (c) 2019, EPFL/Blue Brain Project
 * All rights reserved. Do not distribute without permission.
 *
 * This file is part of Brayns <https://github.com/BlueBrain/Brayns>
 *
 * This library is free software; you can redistribute it and/or modify it under
 * the terms of the GNU Lesser General Public License version 3.0 as published
 * by the Free Software Foundation.
 *
 * This library is distributed in the hope that it will be useful, but WITHOUT
 * ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS
 * FOR A PARTICULAR PURPOSE.  See the GNU Lesser General Public License for more
 * details.
 *
 * You should have received a copy of the GNU Lesser General Public License
 * along with this library; if not, write to the Free Software Foundation, Inc.,
 * 51 Franklin Street, Fifth Floor, Boston, MA 02110-1301 USA.
 */

#include <optix_world.h>

#include "Helpers.h"

struct PerRayData_radiance
{
    float3 result;
    float importance;
    int depth;
};

rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(float3, bgColor, , );
rtTextureSampler<float4, 2> envmap;
rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );
rtDeclareVariable(uint, use_envmap, , );

RT_PROGRAM void envmap_miss()
{
    if (use_envmap)
    {
        float theta = atan2f(ray.direction.x, ray.direction.z);
        float phi = M_PIf * 0.5f - acosf(ray.direction.y);
        float u = (theta + M_PIf) * (0.5f * M_1_PIf);
        float v = -0.5f * (1.0f + sin(phi));
        prd_radiance.result = make_float3(tex2D(envmap, u, v));
    }
    else
    {
        prd_radiance.result = bgColor;
    }
}
