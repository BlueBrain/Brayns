#include "hip/hip_runtime.h"
/* Copyright (c) 2019, EPFL/Blue Brain Project
 * All rights reserved. Do not distribute without permission.
 *
 * This file is part of Brayns <https://github.com/BlueBrain/Brayns>
 *
 * This library is free software; you can redistribute it and/or modify it under
 * the terms of the GNU Lesser General Public License version 3.0 as published
 * by the Free Software Foundation.
 *
 * This library is distributed in the hope that it will be useful, but WITHOUT
 * ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS
 * FOR A PARTICULAR PURPOSE.  See the GNU Lesser General Public License for more
 * details.
 *
 * You should have received a copy of the GNU Lesser General Public License
 * along with this library; if not, write to the Free Software Foundation, Inc.,
 * 51 Franklin Street, Fifth Floor, Boston, MA 02110-1301 USA.
 */

#include "../../CommonStructs.h"
#include "../Helpers.h"
#include "../Random.h"
#include <optix.h>
#include <optixu/optixu_math_namespace.h>

using namespace optix;

rtDeclareVariable(float3, eye, , );
rtDeclareVariable(float3, U, , );
rtDeclareVariable(float3, V, , );
rtDeclareVariable(float3, W, , );
rtDeclareVariable(float3, bad_color, , );
rtDeclareVariable(float, scene_epsilon, , );
rtBuffer<uchar4, 2> output_buffer;
rtBuffer<float4, 2> accum_buffer;
rtDeclareVariable(rtObject, top_object, , );
rtDeclareVariable(unsigned int, radiance_ray_type, , );
rtDeclareVariable(unsigned int, frame, , );
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );

rtDeclareVariable(float, aperture_radius, , );
rtDeclareVariable(float, focal_scale, , );
rtDeclareVariable(float4, jitter4, , );
rtDeclareVariable(unsigned int, samples_per_pixel, , );

rtBuffer<float4, 1> clip_planes;
rtDeclareVariable(unsigned int, nb_clip_planes, , );

__device__ void getClippingValues(const float3& ray_origin,
                                  const float3& ray_direction, float& near,
                                  float& far)
{
    for (int i = 0; i < nb_clip_planes; ++i)
    {
        float4 clipPlane = clip_planes[i];
        const float3 planeNormal = {clipPlane.x, clipPlane.y, clipPlane.z};
        float rn = dot(ray_direction, planeNormal);
        if (rn == 0.f)
            rn = scene_epsilon;
        float d = clipPlane.w;
        float t = -(dot(planeNormal, ray_origin) + d) / rn;
        if (rn > 0.f) // opposite direction plane
            near = max(near, t);
        else
            far = min(far, t);
    }
}

// Pass 'seed' by reference to keep randomness state
__device__ float3 launch(unsigned int& seed, const float2 screen,
                         const bool use_randomness)
{
    // Subpixel jitter: send the ray through a different position inside the
    // pixel each time, to provide antialiasing.
    float2 subpixel_jitter =
        use_randomness ? make_float2(rnd(seed) - 0.5f, rnd(seed) - 0.5f)
                       : make_float2(0.f, 0.f);

    float2 p =
        (make_float2(launch_index) + subpixel_jitter) / screen * 2.f - 1.f;

    // We compute approximate partial derivative according to "Tracing Ray
    // Diffentials by Homan Igehy" paper.
    float3 ray_origin = eye;
    const float fs = focal_scale == 0.f ? 1.f : focal_scale;
    const float3 d = fs * (p.x * U + p.y * V + W);
    const float3 ray_direction = normalize(d);
    const float dotD = dot(d, d);
    const float denom = pow(dotD, 1.5f);

    PerRayData_radiance prd;
    prd.importance = 1.f;
    prd.depth = 0;
    prd.rayDdx = (dotD * U - dot(d, U) * d) / (denom * screen.x);
    prd.rayDdy = (dotD * V - dot(d, V) * d) / (denom * screen.y);

    // lens sampling
    float2 sample = optix::square_to_disk(make_float2(jitter4.z, jitter4.w));

    ray_origin =
        ray_origin +
        aperture_radius * (sample.x * normalize(U) + sample.y * normalize(V));

    float near = scene_epsilon;
    float far = INFINITY;

    getClippingValues(ray_origin, ray_direction, near, far);
    optix::Ray ray(ray_origin, ray_direction, radiance_ray_type, near, far);

    rtTrace(top_object, ray, prd);

    return prd.result;
}

RT_PROGRAM void perspectiveCamera()
{
    const size_t2 screen = output_buffer.size();
    const float2 screen_f = make_float2(screen);

    unsigned int seed =
        tea<16>(screen.x * launch_index.y + launch_index.x, frame);

    const int num_samples = max(1, samples_per_pixel);
    // We enable randomness if we are using subpixel sampling or accumulation
    const bool use_randomness = frame > 0 || num_samples > 1;

    float3 result = make_float3(0, 0, 0);
    for (int i = 0; i < num_samples; i++)
        result += launch(seed, screen_f, use_randomness);
    result /= num_samples;

    float4 acc_val;
    if (frame > 0)
    {
        acc_val = accum_buffer[launch_index];
        acc_val = lerp(acc_val, make_float4(result, 0.f),
                       1.0f / static_cast<float>(frame + 1));
    }
    else
        acc_val = make_float4(result, 1.f);

    output_buffer[launch_index] = make_color(make_float3(acc_val));

    if(accum_buffer.size().x > 1 && accum_buffer.size().y > 1)
        accum_buffer[launch_index] = acc_val;
}

RT_PROGRAM void exception()
{
    output_buffer[launch_index] = make_color(bad_color);
}
