#include "hip/hip_runtime.h"
/* Copyright (c) 2019, EPFL/Blue Brain Project
 * All rights reserved. Do not distribute without permission.
 *
 * This file is part of Brayns <https://github.com/BlueBrain/Brayns>
 *
 * This library is free software; you can redistribute it and/or modify it under
 * the terms of the GNU Lesser General Public License version 3.0 as published
 * by the Free Software Foundation.
 *
 * This library is distributed in the hope that it will be useful, but WITHOUT
 * ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS
 * FOR A PARTICULAR PURPOSE.  See the GNU Lesser General Public License for more
 * details.
 *
 * You should have received a copy of the GNU Lesser General Public License
 * along with this library; if not, write to the Free Software Foundation, Inc.,
 * 51 Franklin Street, Fifth Floor, Boston, MA 02110-1301 USA.
 */

#include "../../CommonStructs.h"
#include "../Helpers.h"
#include "../Random.h"
#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_matrix_namespace.h>

static const float OPENDECK_RADIUS = 2.55f;
static const float OPENDECK_HEIGHT = 2.3f;
static const float OPENDECK_METALSTRIPE_HEIGHT = 0.045f;
static const float PI = 3.141592f;
static const float OPENDECK_BEZEL_ANGLE = PI / 180.0f * 7.98995f;
static const float ANGLE_PER_BORDER_SEGMENT =
    (PI - 8.0f * OPENDECK_BEZEL_ANGLE) / 7.0f + OPENDECK_BEZEL_ANGLE;
static const float FULL_ANGLE = ANGLE_PER_BORDER_SEGMENT + OPENDECK_BEZEL_ANGLE;

using namespace optix;

rtDeclareVariable(unsigned int, segmentID, , ); // even segmentsID are right eye
                                                // buffers and odd are left eye
                                                // buffers
rtDeclareVariable(float3, headPos, , );
rtDeclareVariable(float3, headUVec, , );

rtDeclareVariable(float, HALF_IPD, , );
rtDeclareVariable(float3, eye, , );
rtDeclareVariable(float3, U, , );
rtDeclareVariable(float3, V, , );
rtDeclareVariable(float3, W, , );
rtDeclareVariable(float3, bad_color, , );
rtDeclareVariable(float, scene_epsilon, , );
rtBuffer<uchar4, 2> output_buffer;
rtBuffer<float4, 2> accum_buffer;
rtDeclareVariable(rtObject, top_object, , );
rtDeclareVariable(unsigned int, radiance_ray_type, , );
rtDeclareVariable(unsigned int, frame, , );
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );

rtDeclareVariable(float, aperture_radius, , ); // Unused for now
rtDeclareVariable(float, focal_scale, , );     // Unused for now
rtDeclareVariable(float4, jitter4, , );        // Unused for now
rtDeclareVariable(unsigned int, samples_per_pixel, , );

rtBuffer<float4, 1> clip_planes;
rtDeclareVariable(unsigned int, nb_clip_planes, , );

__device__ void getClippingValues(const float3& ray_origin,
                                  const float3& ray_direction, float& near,
                                  float& far)
{
    for (int i = 0; i < nb_clip_planes; ++i)
    {
        float4 clipPlane = clip_planes[i];
        const float3 planeNormal = {clipPlane.x, clipPlane.y, clipPlane.z};
        float rn = dot(ray_direction, planeNormal);
        if (rn == 0.f)
            rn = scene_epsilon;
        float d = clipPlane.w;
        float t = -(dot(planeNormal, ray_origin) + d) / rn;
        if (rn > 0.f) // opposite direction plane
            near = max(near, t);
        else
            far = min(far, t);
    }
}

// Pass 'seed' by reference to keep randomness state
__device__ float3 launch(unsigned int& seed, const float2 screen,
                         const bool use_randomness)
{
    float eyeDelta = 0.0f;
    float alpha = 0.0f;
    float3 dPx, dPy;

    float2 sample = make_float2(launch_index) / screen;

    if (segmentID <= 13 && segmentID % 2 == 0)
    {
        eyeDelta = HALF_IPD;
        unsigned int angularOffset = segmentID / 2;

        if (segmentID == 0)
            alpha = sample.x * FULL_ANGLE;
        else if (segmentID == 12)
            alpha = PI - FULL_ANGLE + sample.x * FULL_ANGLE;
        else
            alpha = angularOffset * (FULL_ANGLE - OPENDECK_BEZEL_ANGLE) +
                    sample.x * FULL_ANGLE;
    }
    else if (segmentID <= 13 && segmentID % 2 == 1)
    {
        eyeDelta = -HALF_IPD;
        unsigned int angularOffset = segmentID / 2;
        if (segmentID == 1)
            alpha = sample.x * FULL_ANGLE;
        else if (segmentID == 13)
            alpha = PI - FULL_ANGLE + sample.x * FULL_ANGLE;
        else
            alpha = angularOffset * (FULL_ANGLE - OPENDECK_BEZEL_ANGLE) +
                    sample.x * FULL_ANGLE;
    }
    else if (segmentID == 14)
    {
        eyeDelta = HALF_IPD;
    }
    else if (segmentID == 15)
    {
        eyeDelta = -HALF_IPD;
    }

    float3 pixelPos;
    if (segmentID <= 13)
    {
        pixelPos.x = OPENDECK_RADIUS * -cosf(alpha);
        pixelPos.y = OPENDECK_METALSTRIPE_HEIGHT + OPENDECK_HEIGHT * sample.y;
        pixelPos.z = OPENDECK_RADIUS * -sinf(alpha);

        dPx = make_float3(FULL_ANGLE * OPENDECK_RADIUS * sinf(alpha), 0.0f,
                          FULL_ANGLE * OPENDECK_RADIUS * cosf(alpha));
        dPy = make_float3(0.0f, OPENDECK_HEIGHT, 0.0f);
    }
    else if (segmentID > 13)
    {
        pixelPos.x = 2.0f * OPENDECK_RADIUS * (sample.x - 0.5f);
        pixelPos.y = 0.0f;
        pixelPos.z = -OPENDECK_RADIUS * sample.y;

        dPx = make_float3(2.0f * OPENDECK_RADIUS, 0.0f, 0.0f);
        dPy = make_float3(0.0f, 0.0f, -OPENDECK_RADIUS);
    }

    // The tracking model of the 3d glasses is inversed
    // so we need to negate CamDu here.
    const float3 eyeDeltaPos = -headUVec * eyeDelta;

    optix::Matrix3x3 transform;
    transform.setCol(0, U);
    transform.setCol(1, V);
    transform.setCol(2, W);

    const float3 d = pixelPos - headPos + eyeDeltaPos;
    const float dotD = dot(d, d);
    const float denom = pow(dotD, 1.5f);
    float3 dir = normalize(d);

    float3 dirDx = (dotD * dPx - dot(d, dPx) * d) / (denom * screen.x);
    float3 dirDy = (dotD * dPy - dot(d, dPy) * d) / (denom * screen.y);

    PerRayData_radiance prd;
    prd.importance = 1.f;
    prd.depth = 0;
    prd.rayDdx = transform * dirDx;
    prd.rayDdy = transform * dirDy;
    dir = transform * dir;

    const float3 org = eye + headPos - eyeDeltaPos;
    float near = scene_epsilon;
    float far = INFINITY;

    getClippingValues(org, dir, near, far);
    optix::Ray ray(org, dir, radiance_ray_type, near, far);

    rtTrace(top_object, ray, prd);

    return prd.result;
}

RT_PROGRAM void openDeckCamera()
{
    const size_t2 screen = output_buffer.size();
    const float2 screen_f = make_float2(screen);

    unsigned int seed =
        tea<16>(screen.x * launch_index.y + launch_index.x, frame);

    const int num_samples = max(1, samples_per_pixel);
    // We enable randomness if we are using subpixel sampling or accumulation
    const bool use_randomness = frame > 0 || num_samples > 1;

    float3 result = make_float3(0, 0, 0);
    for (int i = 0; i < num_samples; i++)
        result += launch(seed, screen_f, use_randomness);
    result /= num_samples;

    float4 acc_val;
    if (frame > 0)
    {
        acc_val = accum_buffer[launch_index];
        acc_val = lerp(acc_val, make_float4(result, 0.f),
                       1.0f / static_cast<float>(frame + 1));
    }
    else
        acc_val = make_float4(result, 1.f);

    output_buffer[launch_index] = make_color(make_float3(acc_val));

    if (accum_buffer.size().x > 1 && accum_buffer.size().y > 1)
        accum_buffer[launch_index] = acc_val;
}

RT_PROGRAM void exception()
{
    output_buffer[launch_index] = make_color(bad_color);
}
